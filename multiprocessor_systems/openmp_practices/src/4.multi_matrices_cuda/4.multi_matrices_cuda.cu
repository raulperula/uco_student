#include "hip/hip_runtime.h"
/**
 * @file    4.multi_matrices_cuda.c
 * @brief   Brief description.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license GPL v3
 * @version 1.0.0
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <ctime>
#include <omp.h>
#include "cutil_inline.h"

// includes, kernels
#include <matrixMul_kernel.cu>

#define F 2000
#define C 2000

void printDiff(float * data1, float * data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d with error.", iListLength);
    printf(" Differences > %.6f...\n", fListTol);
    int i, j, k;
    int error_count = 0;
    for(j=0;j<height;j++){
        for(i=0;i<width;i++){
            k = j*width+i;
            float fDiff = fabs(data1[k]-data2[k]);
            if(fDiff>fListTol){
                if(error_count < iListLength){
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d of %d\n\n", error_count, WA*HA);
}

// matrix multiplication in cpu
void matrixMulHOMP(float * C, const float * A, const float * B, unsigned int hA, unsigned int wA, unsigned int wB)
{
	#pragma omp parallel for shared(C,suma,i,j,k)
	for(unsigned int i=0;i<hA;++i){
		for(unsigned int j=0;j<wB;++j){
			double sum = 0;
			for(unsigned int k=0;k<wA;++k){
				sum += A[i*wA+k]*B[k*wB+j];
			}
			C[i*wB+j] = (float) sum;
		}
	}
}
void matrixMulH(float * C, const float * A, const float * B, unsigned int hA, unsigned int wA, unsigned int wB)
{
	for(unsigned int i=0;i<hA;++i){
		for(unsigned int j=0;j<wB;++j){
			double sum = 0;
			for(unsigned int k=0;k<wA;++k){
				sum += A[i*wA+k]*B[k*wB+j];
			}
			C[i*wB+j] = (float) sum;
		}
	}
}

// Allocates a matrix with random float entries.
void randomInit(float * data, int size)
{
    for(long int i=0;i<size;++i){
		data[i] = rand()/(float)RAND_MAX;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    unsigned int size_B, mem_size_B, cmem_size_B;

    // set seed for rand()
    srand(1988);

    // allocate host memory for matrices A and B
    // check if padding is active (option == 1)
    unsigned int size_A = WA*HA;
	unsigned int mem_size_A = sizeof(float)*size_A;

	float * h_A = (float *) malloc(mem_size_A);

	size_B = WB*HB;
	mem_size_B = sizeof(float)*size_B;

	float * h_B = (float *) malloc(mem_size_B);

	//~ printf("\nA size: %dx%d\n", WA, HA);
	//~ printf("B size: %dx%d\n", WB, HB);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float * d_A;
    cutilSafeCall(hipMalloc((void **) &d_A, mem_size_A));

	float * d_B;
	cutilSafeCall(hipMalloc((void **) &d_B, mem_size_B));

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice));

    // allocate device memory for result
    unsigned int size_C = WC*HC;
    unsigned int mem_size_C = sizeof(float)*size_C;

    float * d_C;
    cutilSafeCall(hipMalloc((void **) &d_C, mem_size_C));

    // allocate host memory for the result
    float * h_C = (float*) malloc(mem_size_C);

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);  // threads/block
    dim3 grid(WC/threads.x+((WC%BLOCK_SIZE == 0)?0:1), HC/threads.y+((HC%BLOCK_SIZE == 0)?0:1)); // total blocks
    
    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

	///////////////////////////////
    // execute the kernel
    ///////////////////////////////
	matrixMulD<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    hipDeviceSynchronize();

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    double dSeconds = cutGetTimerValue(timer)/(1000.0);
    cutilCheckError(cutDeleteTimer(timer));

	printf("\nDevice time\t= %.8f s\n", dSeconds);

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost));

    // compute reference solution
    float * hc_C = (float *) malloc(mem_size_C);

	///////////////////////////////
    // execute host multiplication
    ///////////////////////////////
    unsigned int timerH = 0;
    cutilCheckError(cutCreateTimer(&timerH));
    cutilCheckError(cutStartTimer(timerH));

	// secuential multiplication
	matrixMulH(hc_C, h_A, h_B, HA, WA, WB);

    cutilCheckError(cutStopTimer(timerH));
    double hSeconds = cutGetTimerValue(timerH)/(1000.0);
	cutilCheckError(cutDeleteTimer(timerH));
	
	printf("\nHost time\t= %.8f s\n", hSeconds);

    // check result
	//~ printDiff(hc_C, h_C, WC, HC, 100, 1.0e-2f);
	
	///////////////////////////////
    // execute OPM multiplication
    ///////////////////////////////
    double t_inicio;
    t_inicio = omp_get_wtime();
    
    // OPM multiplication
	matrixMulHOMP(hc_C, h_A, h_B, HA, WA, WB);
	
	printf("\nOMP time\t= %.8f s\n", omp_get_wtime()-t_inicio);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(hc_C);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}
