#include "hip/hip_runtime.h"
/**
 * @file    vectoresCUDA2.cu
 * @brief   CUDA program.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 *
 * @use How to compile:
 *
 * nvcc vectoresCuda.cu -o vectoresCuda
 * -O (1,2,3) para optimización de código
 */

#include <iostream>
#include <time.h>
// se necesita incluir esta biblioteca para usar CUDA
#include <hip/hip_runtime.h>
#include <common_functions.h>

using namespace std;

void checkCUDAError(const char *msg);
void procesarConCPU(int arrayOrigen[], int arrayDestino[], int tamVector);
__global__ void d_procesarVectores(hipDeviceptr_t arrayOrigen, hipDeviceptr_t arrayDestino, int tamVector);
void procesarConGPU(int * arrayOrigen, int * arrayDestino, int tamVector);

int main(int argc, char* argv[])
{
    const int tamVector = 25000000; // Los vectores tienen 25M de elementos
    clock_t tmp;

    // se crean los dos vectores (dinamicos)
    int * arrayOrigen = new int[tamVector];
    int * arrayDestino = new int[tamVector];

    // se inicializa el primero
    for(int i=0; i<tamVector; i++) {
        arrayOrigen[i] = i;
    }

    // se mide cuánto se tarda en procesar los vectores con la CPU
    tmp = clock();
    procesarConCPU(arrayOrigen, arrayDestino, tamVector);
    clock_t tiempoConCPU = clock()-tmp;

    for(int i=0;i<10;i++){
        cout<<arrayDestino[i]<<' ';
    }
    cout<<endl;

    // se mide cuánto se tarda en procesar los vectores con la GPU
    tmp = clock();
    procesarConGPU(arrayOrigen, arrayDestino, tamVector);
    clock_t tiempoConGPU = clock()-tmp;

    for(int i=0;i<10;i++){
        cout<<arrayDestino[i]<<' ';
    }
    cout<<endl;

    //~ cout << "Los tiempos de procesamiento para "
        //~ << tamVector << " elementos son: " << endl;
    //~ cout << " * CPU: "
        //~ << ((float) tiempoConCPU)/CLOCKS_PER_SEC
        //~ << " segundos" << endl;
    //~ cout << " * GPU: "
        //~ << ((float) tiempoConGPU)/CLOCKS_PER_SEC
        //~ << " segundos" << endl;
    //~ cout << "La GPU ha sido " << ((float) tiempoConCPU)/tiempoConGPU
        //~ << " veces más rápida." << endl;
}

// procesandolo con la CPU en un bucle
// se asigna al elemento i de arrayDestino la suma de los valores de las
// posiciones i e i+1 en arrayOrigen, elevado al cubo.
void procesarConCPU(int arrayOrigen[], int arrayDestino[], int tamVector)
{
    for(int i=0; i<tamVector; i++){
        arrayDestino[i] = (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector])
                        * (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector])
                        * (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector]);
    }
}

// procesandolo con CUDA: invocando un kernel
void procesarConGPU(int * arrayOrigen, int * arrayDestino, int tamVector)
{
    // la GPU trabaja sobre distinta RAM: se reserva memoria y se copian allí los datos.
    // poniendo el prefijo d_ nos ayuda a diferenciar los datos que están en el device (la GPU)
    hipDeviceptr_t d_arrayOrigen;
    hipDeviceptr_t d_arrayDestino;

    // hipMalloc reserva la memoria y asigna el puntero al valor correcto
    hipMalloc((void **) &d_arrayOrigen, sizeof(int)*tamVector);
    checkCUDAError("hipMalloc(l.92)");
    hipMalloc((void **) &d_arrayDestino, sizeof(int)*tamVector);
    checkCUDAError("hipMalloc(l.94)");

    // hipMemcpy copia los datos desde la RAM normal a la de la GPU
        // el primer argumento es la zona de memoria de destino
        // el segundo argumento es la zona de memoria de origen
        // el tercer argumento es el tamaño en bytes a copiar
        // el cuarto argumento es la dirección en la que circulan los datos
    hipMemcpyHtoD(d_arrayOrigen, arrayOrigen, sizeof(int)*tamVector);
    checkCUDAError("hipMemcpy(l.98)");

    // esta llamada invoca al kernel, que se ejecuta en la GPU a la vez en múltiples
    // tareas organizadas en bloques
    int tamBloque = 100;
    int nBloques = tamVector/tamBloque;

    d_procesarVectores <<<tamBloque,nBloques>>> (d_arrayOrigen, d_arrayDestino, tamVector);

    // hipMemcpy espera a que todos los kernels se hayan terminado de ejecutar
    // y copia de vuelta los datos procesados. Ahora la dirección de los datos
    // ha cambiado.
    hipMemcpyDtoH(arrayDestino, d_arrayDestino, sizeof(int)*tamVector);
    checkCUDAError("hipMemcpy(l.111)");

    // se libera la memoria de la GPU
    //~ hipFree(d_arrayOrigen);
    //~ checkCUDAError("hipFree(l.)");
    //~ hipFree(d_arrayDestino);
    //~ checkCUDAError("hipFree(l.)");
}

// kernel que se ejecuta en paralelo en múltiples tareas para un vector
__global__ void d_procesarVectores(hipDeviceptr_t arrayOrigen, hipDeviceptr_t arrayDestino, int tamVector)
{
    // cada tarea tiene un ID numérico basado en su posición en el bloque
    int i = blockIdx.x*blockDim.x+threadIdx.x;

    // cada tarea trabaja sobre una porción de los datos
    arrayDestino[i] = (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector])
                    * (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector])
                    * (arrayOrigen[i]+arrayOrigen[(i+1)%tamVector]);
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if(hipSuccess != err){
        cerr<<"Cuda error: "<<msg<<": "<<hipGetErrorString(err)<<endl;
        //~ exit(-1);
    }
}
