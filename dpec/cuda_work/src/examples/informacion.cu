/**
 * @file    matrixMul_kernel.cu
 * @brief   CUDA program to get information of the GPU device.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 */

#include <iostream>

using namespace std;

int main()
{
    hipDeviceProp_t * prop;

    int count;

    hipGetDeviceCount(&count); // se cuenta el numero de devices
    cout<<"num: "<<count<<endl;

    for(int i=0;i<count;i++){
        hipGetDeviceProperties(prop,i); // se obtienen las propiedades del device
        cout<<"\nGeneral Information for device----"<<i<<endl;
        cout<<"Name "<<prop->name<<endl;
        cout<<"Compute capability "<<prop->major<<". "<<prop->minor<<endl;
        cout<<"Clock Rate "<<prop->clockRate<<endl;
        cout<<"Device copy overlap:  ";

        if(prop->deviceOverlap)
            cout<<"Enable\n";
        else
            cout<<"Disabled\n";

        if(prop->kernelExecTimeoutEnabled)
            cout<<"Enable\n";
        else
            cout<<"Disabled\n";

        cout<<"---Memory Information for device "<<i<<endl;
        cout<<"Total global Mem: "<<prop->totalGlobalMem<<endl;
        cout<<"Total COnstant Mem: "<<prop->totalConstMem<<endl;
        cout<<"Max mem pitch: "<<prop->memPitch<<endl;
        cout<<"Texture ALignment "<<prop->textureAlignment;
        cout<<"\n -------MP Information for devices "<<i;
        cout<<" \nMultiprocessor count : "<<prop->multiProcessorCount<<endl;
        cout<<"Shared mem per mp: "<<prop->sharedMemPerBlock<<endl;
        cout<<"Registers per mp: "<<prop->regsPerBlock<<endl;
        cout<<"Threads in warp: "<<prop->warpSize<<endl;
        cout<<"Max Threads per block "<<prop->maxThreadsPerBlock<<endl;
        cout<<"Max Threads dimensions: ("<<prop->maxThreadsDim[0]<<","<<prop->maxThreadsDim[1]
            <<","<<prop->maxThreadsDim[2]<<")\n";

        cout<<"Max grid dimensions: ("<<prop->maxGridSize[0]<<","<<prop->maxGridSize[1]
            <<","<<prop->maxGridSize[2]<<")\n";
        cout<<endl;
    }

    return 0;
}
