#include "hip/hip_runtime.h"
/**
 * @file    matrizCuda.cu
 * @brief   CUDA program.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 */

#include <iostream>

#define TILE_WIDTH 4
using namespace std;

__global__ void d_procesarMatriz(int * d_a, int * d_b, int * d_c, int width);
void checkCUDAError(const char *msg);   // funcion para comprobar los errores de CUDA

int main(int argc, char** argv)
{
    // se crean los punteros de las matrices origen
    int * h_a, * h_b, * h_c;

    // se crean los punteros de las matrices destino
    int * d_a, * d_b, * d_c;

    // se define la rejilla y el tamaño de bloque
    int numBlocks = 4;
    int numThreadsPerBlock = 16;

    // Parte 1 de 5: reservar memoria principal y en GPU
    size_t memSize = sizeof(int)*numBlocks*numThreadsPerBlock;

    h_a = (int *) malloc(memSize);
    hipMalloc((void**) &d_a,memSize );

    h_b = (int *) malloc(memSize);
    hipMalloc((void**) &d_b,memSize );

    h_c = (int *) malloc(memSize);
    hipMalloc((void**) &d_c,memSize );

    for(int n=0;n<numThreadsPerBlock*numBlocks;n++){
        h_a[n] = h_b[n] = 1;
    }

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(numBlocks/2, numBlocks/2 );
    dim3 dimBlock(numThreadsPerBlock/TILE_WIDTH, numThreadsPerBlock/TILE_WIDTH);

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memSize, hipMemcpyHostToDevice);

    MatrixTiles<<< dimGrid ,dimBlock  >>>(d_a,d_b,d_c,TILE_WIDTH*2);

    // block until the device has completed
    hipDeviceSynchronize();
    hipMemcpy(h_c, d_c, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    for(int i=0;i<8;i++){
        for(int j=0;j<8;j++)
            cout<<h_c[8*i+j]<<" ";
        cout<<endl;
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

// kernel que se ejecuta en paralelo en múltiples tareas para una matriz
__global__ void d_procesarMatriz(int * d_a, int * d_b,int * d_c, int width)
{
    int row = blockIdx.x*TILE_WIDTH+threadIdx.x;
    int col = blockIdx.y*TILE_WIDTH+threadIdx.y;

    int pvalue = 0;

    // se calcula para cada uno de los elementos de la submatriz
    for(int i=0;i<width;i++){
        pvalue += (d_a[row*width+i]*d_b[i*width+col]);
    }

    d_c[row*width+col] = pvalue;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if(hipSuccess != err){
        cerr<<"Cuda error: "<<msg<<": "<<hipGetErrorString(err)<<endl;
        exit(-1);
    }
}
