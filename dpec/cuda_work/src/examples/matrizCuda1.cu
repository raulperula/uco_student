#include "hip/hip_runtime.h"
/**
 * @file    matrizCuda1.cu
 * @brief   CUDA program.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 */

#include <iostream>
#include <stdio.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void MatrixSimple(int *d_a ,int *d_b,int *d_c,int width )
{
    int row = blockIdx.x*width+threadIdx.x;
    int col = blockIdx.y*width+threadIdx.y;

    //for each computes one element of the block sub-matrix
    for(int i=0;i<width;i++){
        d_c[row*width+col] = d_a[row*width+i]+d_b[i*width+col];
    }
}

int main( int argc, char** argv)
{
    // pointer for host memory
    int * h_a, * h_b, * h_c;

    // pointer for device memory
    int * d_a, * d_b, * d_c;

    // define grid and block size
    int numBlocks = 1;
    int numThreadsPerBlock = 64;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks*numThreadsPerBlock*sizeof(int);

    h_a = (int *) malloc(memSize);
    if(hipMalloc((void**) &d_a,memSize) != hipSuccess){
        cout<<"ERROR hipMalloc"<<endl;
        exit(-1);
    }
    h_b = (int *) malloc(memSize);
    hipMalloc((void**) &d_b,memSize);
    h_c = (int *) malloc(memSize);
    hipMalloc((void**) &d_c,memSize);

    for(int n=0;n<numThreadsPerBlock*numBlocks;n++){
        h_a[n] = h_b[n] = 1;
        h_c[n] = 0;
    }

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock/8, numThreadsPerBlock/8);

    // check if kernel execution generated an error
    //~ checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memSize, hipMemcpyHostToDevice);

    MatrixSimple <<< dimGrid,dimBlock >>>(d_a, d_b, d_c, numThreadsPerBlock/8);

    // block until the device has completed
    hipDeviceSynchronize();
    hipMemcpy(h_c, d_c, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    //~ checkCUDAError("hipMemcpy");

    for(int i=0;i<8;i++){
        for(int j=0;j<8;j++)
            cout<<h_c[8*i+j]<<"\t";
        cout<<endl;
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors. Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}
