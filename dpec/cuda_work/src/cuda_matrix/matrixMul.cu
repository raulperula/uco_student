#include "hip/hip_runtime.h"
/**
 * @file    matrixMul.cu
 * @brief   CUDA program.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <ctime>
#include "cutil_inline.h"

// includes, kernels
#include <matrixMul_kernel.cu>

void printDiff(float * data1, float * data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d with error.", iListLength);
    printf(" Differences > %.6f...\n", fListTol);
    int i, j, k;
    int error_count = 0;
    for(j=0;j<height;j++){
        for(i=0;i<width;i++){
            k = j*width+i;
            float fDiff = fabs(data1[k]-data2[k]);
            if(fDiff>fListTol){
                if(error_count < iListLength){
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d of %d\n\n", error_count, WA*HA);
}

// matrix multiplication in cpu
void matrixMulH(int option, float * C, const float * A, const float * B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    if(option == 0 || option == 1){
        // normal or padding execution
        for(unsigned int i=0;i<hA;++i){
            for(unsigned int j=0;j<wB;++j){
                double sum = 0;
                for(unsigned int k=0;k<wA;++k){
                    sum += A[i*wA+k]*B[k*wB+j];
                }
                C[i*wB+j] = (float) sum;
            }
        }
    }
    else if(option == 2){
        // tiling execution

        // Cache L1: C = 64, W = 8
        // Cache L2: C = 4092, W = 8

        // i -> num. iter. / W
        // j -> C(L1) / beta
        // k -> C(L1) / beta * num. appearances
        int TTI = 2000/8;
        int TTJ = 64/2;
        int TTK = 64/(2*2);

        for(unsigned int ii=0;ii<2000;ii+=TTI){
            for(unsigned int kk=0;kk<2000;kk+=TTK){
                for(unsigned int jj=0;jj<2000;jj+=TTJ){
                    int mini = MIN(ii+TTI,2000);
                    for(unsigned int i=ii;i<mini;i++){
                        int mink = MIN(kk+TTK,2000);
                        for(unsigned int k=kk;k<mink;k++){
                            int minj = MIN(jj+TTJ,2000);
                            for(unsigned int j=jj;j<minj;j++){
                                C[i*wB+j] += A[i*wA+k]*B[k*wB+j];
                            }
                        }
                    }
                }
            }
        }
    }
    else{
        printf("\nOpción incorrecta\n");
    }
}

// Allocates a matrix with random float entries.
void randomInit(float * data, int size)
{
    for(long int i=0;i<size;++i){
        data[i] = rand()/(float)RAND_MAX;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    unsigned int size_B, mem_size_B, cmem_size_B;
    int option;

    if(argc == 2){
        option = atoi(argv[1]);
    }
    else{
        option = 0;
    }

    // set seed for rand()
    srand(1988);

    // allocate host memory for matrices A and B
    // check if padding is active (option == 1)
    unsigned int size_A = WA*HA;
    unsigned int mem_size_A = sizeof(float)*size_A;

    float * h_A = (float *) malloc(mem_size_A);

    if(option == 1){ // if option is 1, apply padding
        size_B = (WB+8)*HB;
        unsigned int csize_B = WB*HB;
        mem_size_B = sizeof(float)*size_B;
        cmem_size_B = sizeof(float)*csize_B;

        printf("\nPadding option: add 8 to B matrix weight\n");
    }
    else{
        size_B = WB*HB;
        mem_size_B = sizeof(float)*size_B;
    }
    float * h_B = (float *) malloc(mem_size_B);

    printf("\nA size: %dx%d\n", WA, HA);
    printf("B size: %dx%d\n", WB, HB);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float * d_A;
    cutilSafeCall(hipMalloc((void **) &d_A, mem_size_A));

    float * d_B;
    if(option == 1){
        cutilSafeCall(hipMalloc((void **) &d_B, cmem_size_B));
    }
    else{
        cutilSafeCall(hipMalloc((void **) &d_B, mem_size_B));
    }

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice));
    if(option == 1){
        cutilSafeCall(hipMemcpy(d_B, h_B, cmem_size_B,hipMemcpyHostToDevice));
    }
    else{
        cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice));
    }

    // allocate device memory for result
    unsigned int size_C = WC*HC;
    unsigned int mem_size_C = sizeof(float)*size_C;

    float * d_C;
    cutilSafeCall(hipMalloc((void **) &d_C, mem_size_C));

    // allocate host memory for the result
    float * h_C = (float*) malloc(mem_size_C);

    // setup execution parameters
    //~ dim3 threads(BLOCK_SIZE, BLOCK_SIZE);  // threads/block
    dim3 threads(32, 32); // threads/block
    //~ dim3 grid(WC/threads.x+((WC%BLOCK_SIZE == 0)?0:1), HC/threads.y+((HC%BLOCK_SIZE == 0)?0:1)); // total blocks
    dim3 grid(4, 4); // total blocks

    printf("\nBlock size: %d\nGrid size: %d, %d\n", BLOCK_SIZE, WC/threads.x, HC/threads.y);

    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    ///////////////////////////////
    // execute the kernel
    ///////////////////////////////
    matrixMulD<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    hipDeviceSynchronize();

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    double dSeconds = cutGetTimerValue(timer)/(1000.0);
    cutilCheckError(cutDeleteTimer(timer));

    //~ double dNumOps = 2.0*(double)WA*(double)HA*(double)WB;
    //~ double gflops = 1.0e-9*dNumOps/(dSeconds/(1000.0));

    //Log througput, etc
    //~ printf("Performance = %.4f GFlop/s\nTime = %f s\nSize = %.0f ops\nWorkgroup = %u\n",
        //~ gflops, dSeconds, dNumOps, threads.x * threads.y);

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost));

    // compute reference solution
    float * hc_C = (float *) malloc(mem_size_C);

    ///////////////////////////////
    // execute host multiplication
    ///////////////////////////////
    unsigned int timerH = 0;
    cutilCheckError(cutCreateTimer(&timerH));
    cutilCheckError(cutStartTimer(timerH));

    // option: 0 - normal, 1 - padding, 2 - tiling
    matrixMulH(option, hc_C, h_A, h_B, HA, WA, WB);

    cutilCheckError(cutStopTimer(timerH));
    double hSeconds = cutGetTimerValue(timerH)/(1000.0);
    cutilCheckError(cutDeleteTimer(timerH));

    printf("\nHost time\t= %.8f s\nDevice time\t= %.8f s\n\n", hSeconds, dSeconds);

    // check result
    printDiff(hc_C, h_C, WC, HC, 100, 1.0e-2f);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(hc_C);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}
