#include "hip/hip_runtime.h"
/**
 * @file    matrixMul.cu
 * @brief   CUDA program.
 *
 * @author  Raul Perula-Martinez <raules@gmail.com>
 * @date    2014-11
 *
 * @license  GPL v3
 * @version 1.0.0
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <cstring>
#include "cutil_inline.h"

// includes, kernels
#include <matrixMul_kernel.cu>

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;
    for (j = 0; j < height; j++) {
        if (error_count < iListLength) {
            //~ printf("\n  Row %d:\n", j);
        }
        for (i = 0; i < width; i++) {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);
            if (fDiff > fListTol) {
                if (error_count < iListLength) {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d\n\n", error_count);
}

void computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;

    float* h_A = (float*)malloc(mem_size_A);

    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;

    float* h_B = (float*)malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // print out matrix A and B
    //~ for(int i = 0; i < size_A; i++) {
       //~ printf("%f ", h_A[i]);
       //~ if(((i + 1) % WA) == 0)
          //~ printf("\n");
    //~ }
 //~
    //~ printf("\n\nMatrix B\n");
    //~ for(int i = 0; i < size_B; i++) {
       //~ printf("%f ", h_B[i]);
       //~ if(((i + 1) % WB) == 0)
          //~ printf("\n");
    //~ }

    // allocate device memory
    float* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));

    float* d_B;
    cutilSafeCall(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice));

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // kernel warmup
    //~ matrixMul<<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
    //~ hipDeviceSynchronize();

    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    hipDeviceSynchronize();

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    double dSeconds = cutGetTimerValue(timer)/(1000.0);
    double dNumOps = 2.0 * (double) WA * (double) HA * (double) WB;
    double gflops = 1.0e-9 * dNumOps/dSeconds;

    //Log througput, etc
    printf("matrixMul, Throughput = %.4f GFlop/s, Time = %.5f s, Size = %.0f Ops, NumDevsUsed = %d, Workgroup = %u\n",
        gflops, dSeconds, dNumOps, 1, threads.x * threads.y);
    cutilCheckError(cutDeleteTimer(timer));

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost));

    // print out the results
    //~ printf("\n\nMatrix C (Results)\n");
    //~ for(int i = 0; i < size_C; i++) {
       //~ printf("%f ", h_C[i]);
       //~ if(((i + 1) % WC) == 0)
          //~ printf("\n");
    //~ }
    //~ printf("\n");

    //~ // compute reference solution
    float* reference = (float*)malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);

    // check result
    printDiff(reference, h_C, WC, HC, 100, 1.0e-4f);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_B));
    cutilSafeCall(hipFree(d_C));

    hipDeviceReset();
}
